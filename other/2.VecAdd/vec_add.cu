#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>  //头文件

typedef float FLOAT;
#define USE_UNIX 1

/* get thread id: 1D block and 2D grid ，blockDim.x 内置变量*/ 
#define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x)

/* get block id: 2D grid */
#define get_bid() (blockIdx.x + blockIdx.y * gridDim.x)

/* warm up, start GPU, optional 先将GPU启动起来，因为实际运行的时候遇到GPU代码再启动会耽搁一点点时间，所以可以先将GPU启动起来 */
void warmup();

/* get time stamp */
double get_time(void);

/* host, add  CPU中运行的代码*/
void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N);

/* device function GPU中运行的代码，__global__标志为GPU核函数，返回类型必须是void  */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 1D block */
    int idx = get_tid();

    if (idx < N) z[idx] = z[idx] + y[idx] + x[idx];
}

void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    int i;

    for (i = 0; i < N; i++) z[i] = z[i] + y[i] + x[i];
}

/* a little system programming */
//系统兼容，不同系统求取时间戳
#if USE_UNIX
#include <sys/time.h>
#include <time.h>

double get_time(void)
{
    struct timeval tv;
    double t;

    gettimeofday(&tv, (struct timezone *)0);
    t = tv.tv_sec + (double)tv.tv_usec * 1e-6;

    return t;
}


#else
#include <windows.h>

double get_time(void)
{
    LARGE_INTEGER timer;
    static LARGE_INTEGER fre;
    static int init = 0;
    double t;

    if (init != 1) {
        QueryPerformanceFrequency(&fre);
        init = 1;
    }

    QueryPerformanceCounter(&timer);

    t = timer.QuadPart * 1. / fre.QuadPart;

    return t;
}
#endif

/* warm up GPU */
__global__ void warmup_knl()
{
    int i, j;

    i = 1;
    j = 2;
    i = i + j;
}

void warmup()
{
    int i;

    for (i = 0; i < 8; i++) {
        warmup_knl<<<1, 256>>>();
    }
}

int main()
{
    int N = 20000000;//这么多个浮点数
    int nbytes = N * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 2D grid */
	//总共的线程数：N + bs - 1  ，再除以块数就是需要的网格数
	//ceil函数的作用是求不小于给定实数的最小整数，网格数只能多不能少
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid = dim3(s, s);


	//申请内存，一个是GPU上的内存d(device),一个是CPU上的内存h(host)
    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL, *hy = NULL;
    FLOAT *dz = NULL, *hz = NULL;

    int itr = 30;
    int i;
    double th, td;

    /* warm up GPU */
    warmup();

    /* allocate GPU mem GPU上申请内存的方式*/
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    if (dx == NULL || dy == NULL || dz == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem CPU上申请内存*/
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    if (hx == NULL || hy == NULL || hz == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %.2f MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
        hz[i] = 1;
    }

    /* copy data to GPU ；把CPU上的数据拷贝到GPU*/
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

    /* warm up */
    warmup();

    /* call GPU */
    hipDeviceSynchronize();//这个函数是使GPU算完了再计算CPU上的，因为GPU和CPU是分开的，激活后就单独运行，CPU也继续运行
    td = get_time();
    
    for (i = 0; i < itr; i++) vec_add<<<grid, bs>>>(dx, dy, dz, N);

    hipDeviceSynchronize();
    td = get_time() - td;

    /* CPU */
    th = get_time();
    for (i = 0; i < itr; i++) vec_add_host(hx, hy, hz, N);
    th = get_time() - th;

    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

	//释放内存
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);

    return 0;
}
